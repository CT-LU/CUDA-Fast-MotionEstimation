#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cutil.h>
#include <cutil_inline.h>
#include "eggMotionEstimation.h"


//---------------------------------------------------------------------------------------------------
short*  pucOrgStart;                                        
short*  pucRefStart;
int     refImageSize;        
int     orgImageSize;

short*  d_yRef;
short*  d_yOrg;




texture<short , 2, hipReadModeElementType> tex_ref;
hipArray* cu_array_ref;
hipChannelFormatDesc channelDesc = hipCreateChannelDesc<short>();
dim3 grid4x4Mv((IMAGE_WIDTH/4), IMAGE_HEIGHT/4);
dim3 grid8x4Mv((IMAGE_WIDTH/8), IMAGE_HEIGHT/4);
dim3 grid4x8Mv((IMAGE_WIDTH/4), IMAGE_HEIGHT/8);
dim3 grid8x8Mv((IMAGE_WIDTH/8), IMAGE_HEIGHT/8);
dim3 grid8x16Mv((IMAGE_WIDTH/8), IMAGE_HEIGHT/16);
dim3 grid16x8Mv((IMAGE_WIDTH/16), IMAGE_HEIGHT/8);
dim3 grid16x16Mv((IMAGE_WIDTH/16), IMAGE_HEIGHT/16);

dim3 gridInterpHpel((IMAGE_WIDTH + SEARCH_RANGE*2)/INTERPOLATEDIM, (IMAGE_HEIGHT + SEARCH_RANGE*2));
dim3 gridInterpQpel(((IMAGE_WIDTH + SEARCH_RANGE*2))/INTERPOLATEDIM, (IMAGE_HEIGHT + SEARCH_RANGE*2));


//---------------------------------------------------------------------------------------------------
__global__ void kernelPmods4x4Mv( int orgStride, short* yOrg,
                                  short* d_mvX, short* d_mvY, uint* d_sad){
        uint tid = threadIdx.x;
        uint bx  = blockIdx.x;
        uint by  = blockIdx.y;
        
        
        short pucCurX = SEARCH_RANGE;
        short pucCurY = SEARCH_RANGE;
        short* pucOrg;


        __shared__ short2 s_mv[BLOCK_DIM];
        __shared__ uint   s_sad[BLOCK_DIM];
        __shared__ short  s_orgMb[16];

        short mvY;
        uint  curSad;

        pucOrg = yOrg + Shift2Next4x4Block(orgStride, by, bx);
        pucCurX += bx*4;
        pucCurY += by*4;
        
        pucCurX += (tid - SEARCH_RANGE);
        
        
        short  pucCurY1;
        short  pucCurY2;
        short  pucCurY3;
        short  pucCurY4;
        short  pucCurY5;
        short  pucCurY6;
        
        uint    cur1Sad = 0;
        uint    cur2Sad = 0;
        uint    cur3Sad = 0;
        uint    cur4Sad = 0;
        uint    cur5Sad = 0;
        uint    cur6Sad = 0;
        

        mvY = 0;
        curSad = 0;
        
       
        pucCurY1  = pucCurY + P1;
        pucCurY2  = pucCurY + P2;
        pucCurY3  = pucCurY + P3;
        pucCurY4  = pucCurY + P4;
        pucCurY5  = pucCurY + P5;
        pucCurY6  = pucCurY + P6;

        if(tid < 16){
                s_orgMb[tid] = pucOrg[Index2DAddress(orgStride, tid>>2, tid&3)];
        }
        __syncthreads();
        
        for(ushort m = 0; m < 4; m++){
                for(ushort n = 0; n < 4; n++){
                        curSad  = __usad(tex2D(tex_ref, pucCurX + n, pucCurY + m), s_orgMb[Index2DAddress(4, m, n)], curSad);
                        cur1Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY1 + m), s_orgMb[Index2DAddress(4, m, n)], cur1Sad);
                        cur2Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY2 + m), s_orgMb[Index2DAddress(4, m, n)], cur2Sad);
                        cur3Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY3 + m), s_orgMb[Index2DAddress(4, m, n)], cur3Sad);
                        cur4Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY4 + m), s_orgMb[Index2DAddress(4, m, n)], cur4Sad);
                        cur5Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY5 + m), s_orgMb[Index2DAddress(4, m, n)], cur5Sad);
                        cur6Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY6 + m), s_orgMb[Index2DAddress(4, m, n)], cur6Sad);
                }
        }

        
        short distance = 0;     // distance is 0 for the central position 
        if(curSad > cur1Sad) { distance = P1; curSad = cur1Sad;}
        if(curSad > cur2Sad) { distance = P2; curSad = cur2Sad;}
        if(curSad > cur3Sad) { distance = P3; curSad = cur3Sad;}
        if(curSad > cur4Sad) { distance = P4; curSad = cur4Sad;}
        if(curSad > cur5Sad) { distance = P5; curSad = cur5Sad;}
        if(curSad > cur6Sad) { distance = P6; curSad = cur6Sad;}
        
        mvY += distance;
        
        for(ushort s = STEP_SIZE; s > 0; s >>= 1){

                pucCurY = pucCurY + distance;
                cur1Sad = 0;
                cur2Sad = 0;
        
                pucCurY1  = pucCurY + (-s);
                pucCurY2  = pucCurY + s;
                for(ushort m = 0; m < 4; m++){
                        for(ushort n = 0; n < 4; n++){
                                
                                cur1Sad  = __usad(tex2D(tex_ref, pucCurX + n, pucCurY1 + m), s_orgMb[Index2DAddress(4, m, n)], cur1Sad);
                                cur2Sad  = __usad(tex2D(tex_ref, pucCurX + n, pucCurY2 + m), s_orgMb[Index2DAddress(4, m, n)], cur2Sad);
                        }
                }

                distance = 0;     // distance is 0 for the central position
                if(curSad > cur1Sad) { distance = -s; curSad = cur1Sad;}
                if(curSad > cur2Sad) { distance = s; curSad = cur2Sad;}
                mvY += distance;          

        }
        
        s_mv[tid].y = mvY;
        s_sad[tid] = curSad;

        s_mv[tid].x = tid - SEARCH_RANGE;
                                


#if SEARCH_RANGE > 32
        __syncthreads();

        if(tid < 64){
                if(s_sad[tid] > s_sad[tid + 64]){
                        s_sad[tid] = s_sad[tid + 64];
                        s_mv[tid].y = s_mv[tid + 64].y;
                        s_mv[tid].x = s_mv[tid + 64].x;
                }
        }
#endif
        __syncthreads();
        
        if(tid < 32){   //in 1 warp, needless __syncthreads()  
                if(s_sad[tid] > s_sad[tid + 32]){
                        s_sad[tid] = s_sad[tid + 32];
                        s_mv[tid].y = s_mv[tid + 32].y;
                        s_mv[tid].x = s_mv[tid + 32].x;
                }
                if(s_sad[tid] > s_sad[tid + 16]){
                        s_sad[tid] = s_sad[tid + 16];
                        s_mv[tid].y = s_mv[tid + 16].y;
                        s_mv[tid].x = s_mv[tid + 16].x;
                }
                if(s_sad[tid] > s_sad[tid + 8]){
                        s_sad[tid] = s_sad[tid + 8];
                        s_mv[tid].y = s_mv[tid + 8].y;
                        s_mv[tid].x = s_mv[tid + 8].x;
                }
                if(s_sad[tid] > s_sad[tid + 4]){
                        s_sad[tid] = s_sad[tid + 4];
                        s_mv[tid].y = s_mv[tid + 4].y;
                        s_mv[tid].x = s_mv[tid + 4].x;
                }
                if(s_sad[tid] > s_sad[tid + 2]){
                        s_sad[tid] = s_sad[tid + 2];
                        s_mv[tid].y = s_mv[tid + 2].y;
                        s_mv[tid].x = s_mv[tid + 2].x;
                }
                if(s_sad[0] > s_sad[1]){
                        d_sad[by*(IMAGE_WIDTH/4) + bx] = s_sad[1];
                        d_mvY[by*(IMAGE_WIDTH/4) + bx] = s_mv[1].y;
                        d_mvX[by*(IMAGE_WIDTH/4) + bx] = s_mv[1].x;
                }else{
                        d_sad[by*(IMAGE_WIDTH/4) + bx] = s_sad[0];
                        d_mvY[by*(IMAGE_WIDTH/4) + bx] = s_mv[0].y;
                        d_mvX[by*(IMAGE_WIDTH/4) + bx] = s_mv[0].x;
                }
        }
}
//---------------------------------------------------------------------------------------------------
__global__ void kernelPmods4x8Mv( int orgStride, short* yOrg,
                                  short* d_mvX, short* d_mvY, uint* d_sad){


        uint tid = threadIdx.x;
        uint bx  = blockIdx.x;
        uint by  = blockIdx.y;
        
      
        short pucCurX = SEARCH_RANGE;
        short pucCurY = SEARCH_RANGE;
        short* pucOrg;
        
         
        __shared__ short2 s_mv[BLOCK_DIM];
        __shared__ uint   s_sad[BLOCK_DIM];
        __shared__ short  s_orgMb[32];
        short mvY;
        uint  curSad;
                        
                        

                           
        pucOrg = yOrg + Shift2Next4x8Block(orgStride, by, bx);
        pucCurX += bx*4;
        pucCurY += by*8;
        
        pucCurX += (tid - SEARCH_RANGE);
        
        short  pucCurY1;
        short  pucCurY2;
        short  pucCurY3;
        short  pucCurY4;
        short  pucCurY5;
        short  pucCurY6;
        
        uint    cur1Sad = 0;
        uint    cur2Sad = 0;
        uint    cur3Sad = 0;
        uint    cur4Sad = 0;
        uint    cur5Sad = 0;
        uint    cur6Sad = 0;
        

        mvY = 0;
        curSad = 0;
        
        
        pucCurY1  = pucCurY + P1;
        pucCurY2  = pucCurY + P2;
        pucCurY3  = pucCurY + P3;
        pucCurY4  = pucCurY + P4;
        pucCurY5  = pucCurY + P5;
        pucCurY6  = pucCurY + P6;

        if(tid < 32){
                s_orgMb[tid] = pucOrg[Index2DAddress(orgStride, tid>>2, tid&3)];
        }
        __syncthreads(); 
        
        for(ushort m = 0; m < 8; m++){
                for(ushort n = 0; n < 4; n++){
                        
                        curSad  = __usad(tex2D(tex_ref, pucCurX + n, pucCurY + m), s_orgMb[Index2DAddress(4, m, n)], curSad);
                        cur1Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY1 + m), s_orgMb[Index2DAddress(4, m, n)], cur1Sad);
                        cur2Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY2 + m), s_orgMb[Index2DAddress(4, m, n)], cur2Sad);
                        cur3Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY3 + m), s_orgMb[Index2DAddress(4, m, n)], cur3Sad);
                        cur4Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY4 + m), s_orgMb[Index2DAddress(4, m, n)], cur4Sad);
                        cur5Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY5 + m), s_orgMb[Index2DAddress(4, m, n)], cur5Sad);
                        cur6Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY6 + m), s_orgMb[Index2DAddress(4, m, n)], cur6Sad);
                }
        }

        
        short distance = 0;    
        if(curSad > cur1Sad) { distance = P1; curSad = cur1Sad;}
        if(curSad > cur2Sad) { distance = P2; curSad = cur2Sad;}
        if(curSad > cur3Sad) { distance = P3; curSad = cur3Sad;}
        if(curSad > cur4Sad) { distance = P4; curSad = cur4Sad;}
        if(curSad > cur5Sad) { distance = P5; curSad = cur5Sad;}
        if(curSad > cur6Sad) { distance = P6; curSad = cur6Sad;}
        
        mvY += distance;
        
        for(ushort s = STEP_SIZE; s > 0; s >>= 1){

                
                pucCurY = pucCurY + distance;
                cur1Sad = 0;
                cur2Sad = 0;
        
                pucCurY1  = pucCurY + (-s);
                pucCurY2  = pucCurY + s;
                for(ushort m = 0; m < 8; m++){
                        for(ushort n = 0; n < 4; n++){
                                
                                cur1Sad  = __usad(tex2D(tex_ref, pucCurX + n, pucCurY1 + m), s_orgMb[Index2DAddress(4, m, n)], cur1Sad);
                                cur2Sad  = __usad(tex2D(tex_ref, pucCurX + n, pucCurY2 + m), s_orgMb[Index2DAddress(4, m, n)], cur2Sad);
                        }
                }

                distance = 0;  
                if(curSad > cur1Sad) { distance = -s; curSad = cur1Sad;}
                if(curSad > cur2Sad) { distance = s; curSad = cur2Sad;}
                mvY += distance;          

        }
        
        s_mv[tid].y = mvY;
        s_sad[tid] = curSad;

        s_mv[tid].x = tid - SEARCH_RANGE;
                                
                       


#if SEARCH_RANGE > 32
        __syncthreads();

        if(tid < 64){
                if(s_sad[tid] > s_sad[tid + 64]){
                        s_sad[tid] = s_sad[tid + 64];
                        s_mv[tid].y = s_mv[tid + 64].y;
                        s_mv[tid].x = s_mv[tid + 64].x;
                }
        }
#endif
        __syncthreads();
        
        if(tid < 32){   //in 1 warp, needless __syncthreads()  
                if(s_sad[tid] > s_sad[tid + 32]){
                        s_sad[tid] = s_sad[tid + 32];
                        s_mv[tid].y = s_mv[tid + 32].y;
                        s_mv[tid].x = s_mv[tid + 32].x;
                }
                if(s_sad[tid] > s_sad[tid + 16]){
                        s_sad[tid] = s_sad[tid + 16];
                        s_mv[tid].y = s_mv[tid + 16].y;
                        s_mv[tid].x = s_mv[tid + 16].x;
                }
                if(s_sad[tid] > s_sad[tid + 8]){
                        s_sad[tid] = s_sad[tid + 8];
                        s_mv[tid].y = s_mv[tid + 8].y;
                        s_mv[tid].x = s_mv[tid + 8].x;
                }
                if(s_sad[tid] > s_sad[tid + 4]){
                        s_sad[tid] = s_sad[tid + 4];
                        s_mv[tid].y = s_mv[tid + 4].y;
                        s_mv[tid].x = s_mv[tid + 4].x;
                }
                if(s_sad[tid] > s_sad[tid + 2]){
                        s_sad[tid] = s_sad[tid + 2];
                        s_mv[tid].y = s_mv[tid + 2].y;
                        s_mv[tid].x = s_mv[tid + 2].x;
                }
                if(s_sad[0] > s_sad[1]){
                        d_sad[by*(IMAGE_WIDTH/4) + bx] = s_sad[1];
                        d_mvY[by*(IMAGE_WIDTH/4) + bx] = s_mv[1].y;
                        d_mvX[by*(IMAGE_WIDTH/4) + bx] = s_mv[1].x;
                }else{
                        d_sad[by*(IMAGE_WIDTH/4) + bx] = s_sad[0];
                        d_mvY[by*(IMAGE_WIDTH/4) + bx] = s_mv[0].y;
                        d_mvX[by*(IMAGE_WIDTH/4) + bx] = s_mv[0].x;
                }
        }
        
}
//---------------------------------------------------------------------------------------------------
__global__ void kernelPmods8x4Mv( int orgStride, short* yOrg,
                                  short* d_mvX, short* d_mvY, uint* d_sad){


        uint tid = threadIdx.x;
        uint bx  = blockIdx.x;
        uint by  = blockIdx.y;
        
        short pucCurX = SEARCH_RANGE;
        short pucCurY = SEARCH_RANGE;
        short* pucOrg;
        
                        
        __shared__ short2 s_mv[BLOCK_DIM];
        __shared__ uint   s_sad[BLOCK_DIM];
        __shared__ short  s_orgMb[32];
        short mvY;
        uint  curSad;
                     
        pucOrg = yOrg + Shift2Next8x4Block(orgStride, by, bx);
        pucCurX += bx*8;
        pucCurY += by*4;
        
        pucCurX += (tid - SEARCH_RANGE);
        
        short  pucCurY1;
        short  pucCurY2;
        short  pucCurY3;
        short  pucCurY4;
        short  pucCurY5;
        short  pucCurY6;
        
        uint    cur1Sad = 0;
        uint    cur2Sad = 0;
        uint    cur3Sad = 0;
        uint    cur4Sad = 0;
        uint    cur5Sad = 0;
        uint    cur6Sad = 0;
        

        mvY = 0;
        curSad = 0;

        pucCurY1  = pucCurY + P1;
        pucCurY2  = pucCurY + P2;
        pucCurY3  = pucCurY + P3;
        pucCurY4  = pucCurY + P4;
        pucCurY5  = pucCurY + P5;
        pucCurY6  = pucCurY + P6;

        if(tid < 32){
                s_orgMb[tid] = pucOrg[Index2DAddress(orgStride, tid>>3, tid&7)];
        }
        __syncthreads();
        
        for(int m = 0; m < 4; m++){
                for(int n = 0; n < 8; n++){
                        
                        curSad  = __usad(tex2D(tex_ref, pucCurX + n, pucCurY + m), s_orgMb[Index2DAddress(8, m, n)], curSad);
                        cur1Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY1 + m), s_orgMb[Index2DAddress(8, m, n)], cur1Sad);
                        cur2Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY2 + m), s_orgMb[Index2DAddress(8, m, n)], cur2Sad);
                        cur3Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY3 + m), s_orgMb[Index2DAddress(8, m, n)], cur3Sad);
                        cur4Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY4 + m), s_orgMb[Index2DAddress(8, m, n)], cur4Sad);
                        cur5Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY5 + m), s_orgMb[Index2DAddress(8, m, n)], cur5Sad);
                        cur6Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY6 + m), s_orgMb[Index2DAddress(8, m, n)], cur6Sad);
                        
                }
        }

        
        short distance = 0;   
        if(curSad > cur1Sad) { distance = P1; curSad = cur1Sad;}
        if(curSad > cur2Sad) { distance = P2; curSad = cur2Sad;}
        if(curSad > cur3Sad) { distance = P3; curSad = cur3Sad;}
        if(curSad > cur4Sad) { distance = P4; curSad = cur4Sad;}
        if(curSad > cur5Sad) { distance = P5; curSad = cur5Sad;}
        if(curSad > cur6Sad) { distance = P6; curSad = cur6Sad;}
        
        mvY += distance;
        
        for(int s = STEP_SIZE; s > 0; s >>= 1){

                
                pucCurY = pucCurY + distance;
                cur1Sad = 0;
                cur2Sad = 0;
        
                
                pucCurY1  = pucCurY + (-s);
                pucCurY2  = pucCurY + s;
                for(int m = 0; m < 4; m++){
                        for(int n = 0; n < 8; n++){
                                
                                cur1Sad  = __usad(tex2D(tex_ref, pucCurX + n, pucCurY1 + m), s_orgMb[Index2DAddress(8, m, n)], cur1Sad);
                                cur2Sad  = __usad(tex2D(tex_ref, pucCurX + n, pucCurY2 + m), s_orgMb[Index2DAddress(8, m, n)], cur2Sad);
                                
                        }
                }

                distance = 0;   
                if(curSad > cur1Sad) { distance = -s; curSad = cur1Sad;}
                if(curSad > cur2Sad) { distance = s; curSad = cur2Sad;}
                mvY += distance;          

        }
        
        s_mv[tid].y = mvY;
        s_sad[tid] = curSad;

        s_mv[tid].x = tid - SEARCH_RANGE;
                                
                        


#if SEARCH_RANGE > 32
        __syncthreads();

        if(tid < 64){
                if(s_sad[tid] > s_sad[tid + 64]){
                        s_sad[tid] = s_sad[tid + 64];
                        s_mv[tid].y = s_mv[tid + 64].y;
                        s_mv[tid].x = s_mv[tid + 64].x;
                }
        }
#endif
        __syncthreads();
        
        if(tid < 32){   //in 1 warp, needless __syncthreads() 
                if(s_sad[tid] > s_sad[tid + 32]){
                        s_sad[tid] = s_sad[tid + 32];
                        s_mv[tid].y = s_mv[tid + 32].y;
                        s_mv[tid].x = s_mv[tid + 32].x;
                }
                if(s_sad[tid] > s_sad[tid + 16]){
                        s_sad[tid] = s_sad[tid + 16];
                        s_mv[tid].y = s_mv[tid + 16].y;
                        s_mv[tid].x = s_mv[tid + 16].x;
                }
                if(s_sad[tid] > s_sad[tid + 8]){
                        s_sad[tid] = s_sad[tid + 8];
                        s_mv[tid].y = s_mv[tid + 8].y;
                        s_mv[tid].x = s_mv[tid + 8].x;
                }
                if(s_sad[tid] > s_sad[tid + 4]){
                        s_sad[tid] = s_sad[tid + 4];
                        s_mv[tid].y = s_mv[tid + 4].y;
                        s_mv[tid].x = s_mv[tid + 4].x;
                }
                if(s_sad[tid] > s_sad[tid + 2]){
                        s_sad[tid] = s_sad[tid + 2];
                        s_mv[tid].y = s_mv[tid + 2].y;
                        s_mv[tid].x = s_mv[tid + 2].x;
                }
                if(s_sad[0] > s_sad[1]){
                        d_sad[by*(IMAGE_WIDTH/8) + bx] = s_sad[1];
                        d_mvY[by*(IMAGE_WIDTH/8) + bx] = s_mv[1].y;
                        d_mvX[by*(IMAGE_WIDTH/8) + bx] = s_mv[1].x;
                }else{
                        d_sad[by*(IMAGE_WIDTH/8) + bx] = s_sad[0];
                        d_mvY[by*(IMAGE_WIDTH/8) + bx] = s_mv[0].y;
                        d_mvX[by*(IMAGE_WIDTH/8) + bx] = s_mv[0].x;
                }
        }
        
}
//---------------------------------------------------------------------------------------------------
__global__ void kernelPmods8x8Mv( int orgStride, short* yOrg,
                                  short* d_mvX, short* d_mvY, uint* d_sad){


        uint tid = threadIdx.x;
        uint bx  = blockIdx.x;
        uint by  = blockIdx.y;
        
        short pucCurX = SEARCH_RANGE;
        short pucCurY = SEARCH_RANGE;
        short* pucOrg;
        
                        
        __shared__ short2 s_mv[BLOCK_DIM];
        __shared__ uint   s_sad[BLOCK_DIM];
        __shared__ short  s_orgMb[64];
        short mvY;
        uint  curSad;
        
                       
        pucOrg = yOrg + Shift2Next8x8Block(orgStride, by, bx);
        pucCurX += bx*8;
        pucCurY += by*8;
        
        pucCurX += (tid - SEARCH_RANGE);
        
        
        short  pucCurY1;
        short  pucCurY2;
        short  pucCurY3;
        short  pucCurY4;
        short  pucCurY5;
        short  pucCurY6;
        
        uint    cur1Sad = 0;
        uint    cur2Sad = 0;
        uint    cur3Sad = 0;
        uint    cur4Sad = 0;
        uint    cur5Sad = 0;
        uint    cur6Sad = 0;
        

        mvY = 0;
        curSad = 0;
        
       
        pucCurY1  = pucCurY + P1;
        pucCurY2  = pucCurY + P2;
        pucCurY3  = pucCurY + P3;
        pucCurY4  = pucCurY + P4;
        pucCurY5  = pucCurY + P5;
        pucCurY6  = pucCurY + P6;

        if(tid < 64){
                s_orgMb[tid] = pucOrg[Index2DAddress(orgStride, tid>>3, tid&7)];
        }
        __syncthreads(); 
        
        for(int m = 0; m < 8; m++){
                for(int n = 0; n < 8; n++){
                        
                        curSad  = __usad(tex2D(tex_ref, pucCurX + n, pucCurY + m), s_orgMb[Index2DAddress(8, m, n)], curSad);
                        cur1Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY1 + m), s_orgMb[Index2DAddress(8, m, n)], cur1Sad);
                        cur2Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY2 + m), s_orgMb[Index2DAddress(8, m, n)], cur2Sad);
                        cur3Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY3 + m), s_orgMb[Index2DAddress(8, m, n)], cur3Sad);
                        cur4Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY4 + m), s_orgMb[Index2DAddress(8, m, n)], cur4Sad);
                        cur5Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY5 + m), s_orgMb[Index2DAddress(8, m, n)], cur5Sad);
                        cur6Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY6 + m), s_orgMb[Index2DAddress(8, m, n)], cur6Sad);
                }
        }

       
        short distance = 0;  
        if(curSad > cur1Sad) { distance = P1; curSad = cur1Sad;}
        if(curSad > cur2Sad) { distance = P2; curSad = cur2Sad;}
        if(curSad > cur3Sad) { distance = P3; curSad = cur3Sad;}
        if(curSad > cur4Sad) { distance = P4; curSad = cur4Sad;}
        if(curSad > cur5Sad) { distance = P5; curSad = cur5Sad;}
        if(curSad > cur6Sad) { distance = P6; curSad = cur6Sad;}
        
        mvY += distance;
        
        for(int s = STEP_SIZE; s > 0; s >>= 1){

         
                pucCurY = pucCurY + distance;
                cur1Sad = 0;
                cur2Sad = 0;
        
                pucCurY1  = pucCurY + (-s);
                pucCurY2  = pucCurY + s;
                for(int m = 0; m < 8; m++){
                        for(int n = 0; n < 8; n++){
                                
                                cur1Sad  = __usad(tex2D(tex_ref, pucCurX + n, pucCurY1 + m), s_orgMb[Index2DAddress(8, m, n)], cur1Sad);
                                cur2Sad  = __usad(tex2D(tex_ref, pucCurX + n, pucCurY2 + m), s_orgMb[Index2DAddress(8, m, n)], cur2Sad);
                        }
                }

                distance = 0;     
                if(curSad > cur1Sad) { distance = -s; curSad = cur1Sad;}
                if(curSad > cur2Sad) { distance = s; curSad = cur2Sad;}
                mvY += distance;          

        }
        
        s_mv[tid].y = mvY;
        s_sad[tid] = curSad;

        s_mv[tid].x = tid - SEARCH_RANGE;
                                
                        
#if SEARCH_RANGE > 32
        __syncthreads();

        if(tid < 64){
                if(s_sad[tid] > s_sad[tid + 64]){
                        s_sad[tid] = s_sad[tid + 64];
                        s_mv[tid].y = s_mv[tid + 64].y;
                        s_mv[tid].x = s_mv[tid + 64].x;
                }
        }
#endif
        __syncthreads();
        
        if(tid < 32){   //in 1 warp, needless __syncthreads()   
                if(s_sad[tid] > s_sad[tid + 32]){
                        s_sad[tid] = s_sad[tid + 32];
                        s_mv[tid].y = s_mv[tid + 32].y;
                        s_mv[tid].x = s_mv[tid + 32].x;
                }
                if(s_sad[tid] > s_sad[tid + 16]){
                        s_sad[tid] = s_sad[tid + 16];
                        s_mv[tid].y = s_mv[tid + 16].y;
                        s_mv[tid].x = s_mv[tid + 16].x;
                }
                if(s_sad[tid] > s_sad[tid + 8]){
                        s_sad[tid] = s_sad[tid + 8];
                        s_mv[tid].y = s_mv[tid + 8].y;
                        s_mv[tid].x = s_mv[tid + 8].x;
                }
                if(s_sad[tid] > s_sad[tid + 4]){
                        s_sad[tid] = s_sad[tid + 4];
                        s_mv[tid].y = s_mv[tid + 4].y;
                        s_mv[tid].x = s_mv[tid + 4].x;
                }
                if(s_sad[tid] > s_sad[tid + 2]){
                        s_sad[tid] = s_sad[tid + 2];
                        s_mv[tid].y = s_mv[tid + 2].y;
                        s_mv[tid].x = s_mv[tid + 2].x;
                }
                if(s_sad[0] > s_sad[1]){
                        d_sad[by*(IMAGE_WIDTH/8) + bx] = s_sad[1];
                        d_mvY[by*(IMAGE_WIDTH/8) + bx] = s_mv[1].y;
                        d_mvX[by*(IMAGE_WIDTH/8) + bx] = s_mv[1].x;
                }else{
                        d_sad[by*(IMAGE_WIDTH/8) + bx] = s_sad[0];
                        d_mvY[by*(IMAGE_WIDTH/8) + bx] = s_mv[0].y;
                        d_mvX[by*(IMAGE_WIDTH/8) + bx] = s_mv[0].x;
                }
        }
        
}
//---------------------------------------------------------------------------------------------------
__global__ void kernelPmods8x16Mv( int orgStride, short* yOrg,
                                   short* d_mvX, short* d_mvY, uint* d_sad){


        uint tid = threadIdx.x;
        uint bx  = blockIdx.x;
        uint by  = blockIdx.y;
        
       
        short pucCurX = SEARCH_RANGE;
        short pucCurY = SEARCH_RANGE;
        short* pucOrg;
        
        
        __shared__ short2 s_mv[BLOCK_DIM];
        __shared__ uint   s_sad[BLOCK_DIM];
        __shared__ short  s_orgMb[128];
        short mvY;
        uint  curSad;
        
        pucOrg = yOrg + Shift2Next8x16Block(orgStride, by, bx);
        pucCurX += bx*8;
        pucCurY += by*16;
       
        pucCurX += (tid - SEARCH_RANGE);
        
        
        short  pucCurY1;
        short  pucCurY2;
        short  pucCurY3;
        short  pucCurY4;
        short  pucCurY5;
        short  pucCurY6;
        
        uint    cur1Sad = 0;
        uint    cur2Sad = 0;
        uint    cur3Sad = 0;
        uint    cur4Sad = 0;
        uint    cur5Sad = 0;
        uint    cur6Sad = 0;
        

        mvY = 0;
        curSad = 0;
        
        pucCurY1  = pucCurY + P1;
        pucCurY2  = pucCurY + P2;
        pucCurY3  = pucCurY + P3;
        pucCurY4  = pucCurY + P4;
        pucCurY5  = pucCurY + P5;
        pucCurY6  = pucCurY + P6;

        if(tid < 128){
                s_orgMb[tid] = pucOrg[Index2DAddress(orgStride, tid>>3, tid&7)];
        }
        __syncthreads(); 
        
        for(int m = 0; m < 16; m++){
                for(int n = 0; n < 8; n++){

                        curSad  = __usad(tex2D(tex_ref, pucCurX + n, pucCurY + m), s_orgMb[Index2DAddress(8, m, n)], curSad);
                        cur1Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY1 + m), s_orgMb[Index2DAddress(8, m, n)], cur1Sad);
                        cur2Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY2 + m), s_orgMb[Index2DAddress(8, m, n)], cur2Sad);
                        cur3Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY3 + m), s_orgMb[Index2DAddress(8, m, n)], cur3Sad);
                        cur4Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY4 + m), s_orgMb[Index2DAddress(8, m, n)], cur4Sad);
                        cur5Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY5 + m), s_orgMb[Index2DAddress(8, m, n)], cur5Sad);
                        cur6Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY6 + m), s_orgMb[Index2DAddress(8, m, n)], cur6Sad);
                }
        }

        
        short distance = 0;     
        if(curSad > cur1Sad) { distance = P1; curSad = cur1Sad;}
        if(curSad > cur2Sad) { distance = P2; curSad = cur2Sad;}
        if(curSad > cur3Sad) { distance = P3; curSad = cur3Sad;}
        if(curSad > cur4Sad) { distance = P4; curSad = cur4Sad;}
        if(curSad > cur5Sad) { distance = P5; curSad = cur5Sad;}
        if(curSad > cur6Sad) { distance = P6; curSad = cur6Sad;}
        
        mvY += distance;
        
        for(int s = STEP_SIZE; s > 0; s >>= 1){

                
                pucCurY = pucCurY + distance;
                cur1Sad = 0;
                cur2Sad = 0;
        
                pucCurY1  = pucCurY + (-s);
                pucCurY2  = pucCurY + s;
                for(int m = 0; m < 16; m++){
                        for(int n = 0; n < 8; n++){
                                
                                cur1Sad  = __usad(tex2D(tex_ref, pucCurX + n, pucCurY1 + m), s_orgMb[Index2DAddress(8, m, n)], cur1Sad);
                                cur2Sad  = __usad(tex2D(tex_ref, pucCurX + n, pucCurY2 + m), s_orgMb[Index2DAddress(8, m, n)], cur2Sad);
                        }
                }

                distance = 0;    
                if(curSad > cur1Sad) { distance = -s; curSad = cur1Sad;}
                if(curSad > cur2Sad) { distance = s; curSad = cur2Sad;}
                mvY += distance;          

        }
        
        s_mv[tid].y = mvY;
        s_sad[tid] = curSad;

        s_mv[tid].x = tid - SEARCH_RANGE;
                                
                       
#if SEARCH_RANGE > 32
        __syncthreads();

        if(tid < 64){
                if(s_sad[tid] > s_sad[tid + 64]){
                        s_sad[tid] = s_sad[tid + 64];
                        s_mv[tid].y = s_mv[tid + 64].y;
                        s_mv[tid].x = s_mv[tid + 64].x;
                }
        }
#endif
        __syncthreads();
        
        if(tid < 32){   //in 1 warp, needless __syncthreads()  
                if(s_sad[tid] > s_sad[tid + 32]){
                        s_sad[tid] = s_sad[tid + 32];
                        s_mv[tid].y = s_mv[tid + 32].y;
                        s_mv[tid].x = s_mv[tid + 32].x;
                }
                if(s_sad[tid] > s_sad[tid + 16]){
                        s_sad[tid] = s_sad[tid + 16];
                        s_mv[tid].y = s_mv[tid + 16].y;
                        s_mv[tid].x = s_mv[tid + 16].x;
                }
                if(s_sad[tid] > s_sad[tid + 8]){
                        s_sad[tid] = s_sad[tid + 8];
                        s_mv[tid].y = s_mv[tid + 8].y;
                        s_mv[tid].x = s_mv[tid + 8].x;
                }
                if(s_sad[tid] > s_sad[tid + 4]){
                        s_sad[tid] = s_sad[tid + 4];
                        s_mv[tid].y = s_mv[tid + 4].y;
                        s_mv[tid].x = s_mv[tid + 4].x;
                }
                if(s_sad[tid] > s_sad[tid + 2]){
                        s_sad[tid] = s_sad[tid + 2];
                        s_mv[tid].y = s_mv[tid + 2].y;
                        s_mv[tid].x = s_mv[tid + 2].x;
                }
                if(s_sad[0] > s_sad[1]){
                        d_sad[by*(IMAGE_WIDTH/8) + bx] = s_sad[1];
                        d_mvY[by*(IMAGE_WIDTH/8) + bx] = s_mv[1].y;
                        d_mvX[by*(IMAGE_WIDTH/8) + bx] = s_mv[1].x;
                }else{
                        d_sad[by*(IMAGE_WIDTH/8) + bx] = s_sad[0];
                        d_mvY[by*(IMAGE_WIDTH/8) + bx] = s_mv[0].y;
                        d_mvX[by*(IMAGE_WIDTH/8) + bx] = s_mv[0].x;
                }
        }
}
//---------------------------------------------------------------------------------------------------
__global__ void kernelPmods16x8Mv( int orgStride, short* yOrg,
                                   short* d_mvX, short* d_mvY, uint* d_sad){

        uint tid = threadIdx.x;
        uint bx  = blockIdx.x;
        uint by  = blockIdx.y;
        
        short pucCurX = SEARCH_RANGE;
        short pucCurY = SEARCH_RANGE;
        short* pucOrg;
        
                        
        __shared__ short2 s_mv[BLOCK_DIM];
        __shared__ uint   s_sad[BLOCK_DIM];
        __shared__ short  s_orgMb[128];
        short mvY;
        uint  curSad;
            
        pucOrg = yOrg + Shift2Next16x8Block(orgStride, by, bx);
        pucCurX += bx*16;
        pucCurY += by*8;
        
        pucCurX += (tid - SEARCH_RANGE);
        
        short  pucCurY1;
        short  pucCurY2;
        short  pucCurY3;
        short  pucCurY4;
        short  pucCurY5;
        short  pucCurY6;
        
        uint    cur1Sad = 0;
        uint    cur2Sad = 0;
        uint    cur3Sad = 0;
        uint    cur4Sad = 0;
        uint    cur5Sad = 0;
        uint    cur6Sad = 0;
        

        mvY = 0;
        curSad = 0;
        
        pucCurY1  = pucCurY + P1;
        pucCurY2  = pucCurY + P2;
        pucCurY3  = pucCurY + P3;
        pucCurY4  = pucCurY + P4;
        pucCurY5  = pucCurY + P5;
        pucCurY6  = pucCurY + P6;

        if(tid < 128){
                s_orgMb[tid] = pucOrg[Index2DAddress(orgStride, tid>>4, tid&15)];
        }
        __syncthreads(); 
        
        for(int m = 0; m < 8; m++){
                for(int n = 0; n < 16; n++){
                        
                        curSad  = __usad(tex2D(tex_ref, pucCurX + n, pucCurY + m), s_orgMb[Index2DAddress(16, m, n)], curSad);
                        cur1Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY1 + m), s_orgMb[Index2DAddress(16, m, n)], cur1Sad);
                        cur2Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY2 + m), s_orgMb[Index2DAddress(16, m, n)], cur2Sad);
                        cur3Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY3 + m), s_orgMb[Index2DAddress(16, m, n)], cur3Sad);
                        cur4Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY4 + m), s_orgMb[Index2DAddress(16, m, n)], cur4Sad);
                        cur5Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY5 + m), s_orgMb[Index2DAddress(16, m, n)], cur5Sad);
                        cur6Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY6 + m), s_orgMb[Index2DAddress(16, m, n)], cur6Sad);
                }
        }

        
        short distance = 0;     
        if(curSad > cur1Sad) { distance = P1; curSad = cur1Sad;}
        if(curSad > cur2Sad) { distance = P2; curSad = cur2Sad;}
        if(curSad > cur3Sad) { distance = P3; curSad = cur3Sad;}
        if(curSad > cur4Sad) { distance = P4; curSad = cur4Sad;}
        if(curSad > cur5Sad) { distance = P5; curSad = cur5Sad;}
        if(curSad > cur6Sad) { distance = P6; curSad = cur6Sad;}
        
        mvY += distance;
        
        for(int s = STEP_SIZE; s > 0; s >>= 1){

               
                pucCurY = pucCurY + distance;
                cur1Sad = 0;
                cur2Sad = 0;
        
                pucCurY1  = pucCurY + (-s);
                pucCurY2  = pucCurY + s;
                
                for(int m = 0; m < 8; m++){
                        for(int n = 0; n < 16; n++){
                                cur1Sad  = __usad(tex2D(tex_ref, pucCurX + n, pucCurY1 + m), s_orgMb[Index2DAddress(16, m, n)], cur1Sad);
                                cur2Sad  = __usad(tex2D(tex_ref, pucCurX + n, pucCurY2 + m), s_orgMb[Index2DAddress(16, m, n)], cur2Sad);
                        }
                }

                distance = 0;  
                if(curSad > cur1Sad) { distance = -s; curSad = cur1Sad;}
                if(curSad > cur2Sad) { distance = s; curSad = cur2Sad;}
                mvY += distance;          

        }
        
        s_mv[tid].y = mvY;
        s_sad[tid] = curSad;

        s_mv[tid].x = tid - SEARCH_RANGE;
                                
        
#if SEARCH_RANGE > 32
        __syncthreads();

        if(tid < 64){
                if(s_sad[tid] > s_sad[tid + 64]){
                        s_sad[tid] = s_sad[tid + 64];
                        s_mv[tid].y = s_mv[tid + 64].y;
                        s_mv[tid].x = s_mv[tid + 64].x;
                }
        }
#endif
        __syncthreads();
        
        if(tid < 32){   //in 1 warp, needless __syncthreads()   
                if(s_sad[tid] > s_sad[tid + 32]){
                        s_sad[tid] = s_sad[tid + 32];
                        s_mv[tid].y = s_mv[tid + 32].y;
                        s_mv[tid].x = s_mv[tid + 32].x;
                }
                if(s_sad[tid] > s_sad[tid + 16]){
                        s_sad[tid] = s_sad[tid + 16];
                        s_mv[tid].y = s_mv[tid + 16].y;
                        s_mv[tid].x = s_mv[tid + 16].x;
                }
                if(s_sad[tid] > s_sad[tid + 8]){
                        s_sad[tid] = s_sad[tid + 8];
                        s_mv[tid].y = s_mv[tid + 8].y;
                        s_mv[tid].x = s_mv[tid + 8].x;
                }
                if(s_sad[tid] > s_sad[tid + 4]){
                        s_sad[tid] = s_sad[tid + 4];
                        s_mv[tid].y = s_mv[tid + 4].y;
                        s_mv[tid].x = s_mv[tid + 4].x;
                }
                if(s_sad[tid] > s_sad[tid + 2]){
                        s_sad[tid] = s_sad[tid + 2];
                        s_mv[tid].y = s_mv[tid + 2].y;
                        s_mv[tid].x = s_mv[tid + 2].x;
                }
                if(s_sad[0] > s_sad[1]){
                        d_sad[by*(IMAGE_WIDTH/16) + bx] = s_sad[1];
                        d_mvY[by*(IMAGE_WIDTH/16) + bx] = s_mv[1].y;
                        d_mvX[by*(IMAGE_WIDTH/16) + bx] = s_mv[1].x;
                }else{
                        d_sad[by*(IMAGE_WIDTH/16) + bx] = s_sad[0];
                        d_mvY[by*(IMAGE_WIDTH/16) + bx] = s_mv[0].y;
                        d_mvX[by*(IMAGE_WIDTH/16) + bx] = s_mv[0].x;
                }
        }
        
}
//---------------------------------------------------------------------------------------------------
__global__ void kernelPmods16x16Mv(int orgStride, short* yOrg,
                                   short* d_mvX, short* d_mvY, uint* d_sad){

        uint tid = threadIdx.x;
        uint bx  = blockIdx.x;
        uint by  = blockIdx.y;
        
        
        short pucCurX = SEARCH_RANGE;
        short pucCurY = SEARCH_RANGE;
        short* pucOrg;
        
                        
        __shared__ short2 s_mv[BLOCK_DIM];
        __shared__ uint   s_sad[BLOCK_DIM];
        __shared__ short  s_orgMb[256];
        short mvY;
        uint  curSad;
                          
        pucOrg = yOrg + Shift2Next16x16Block(orgStride, by, bx);

        pucCurX += bx*16;
        pucCurY += by*16;
        
        pucCurX += (tid - SEARCH_RANGE);

        curSad = 0;
    

        short  pucCurY1;
        short  pucCurY2;
        short  pucCurY3;
        short  pucCurY4;
        short  pucCurY5;
        short  pucCurY6;
        
        uint    cur1Sad = 0;
        uint    cur2Sad = 0;
        uint    cur3Sad = 0;
        uint    cur4Sad = 0;
        uint    cur5Sad = 0;
        uint    cur6Sad = 0;
        
        mvY = 0;
        curSad = 0;
        
        pucCurY1  = pucCurY + P1;
        pucCurY2  = pucCurY + P2;
        pucCurY3  = pucCurY + P3;
        pucCurY4  = pucCurY + P4;
        pucCurY5  = pucCurY + P5;
        pucCurY6  = pucCurY + P6;

        
        s_orgMb[tid] = pucOrg[Index2DAddress(orgStride, tid>>4, tid&15)];
        s_orgMb[tid + 128] = pucOrg[Index2DAddress(orgStride, 8 + (tid>>4), tid&15)];
        __syncthreads(); 
        
        for(int m = 0; m < 16; m++){
                for(int n = 0; n < 16; n++){
                        
                        curSad  = __usad(tex2D(tex_ref, pucCurX + n, pucCurY + m), s_orgMb[Index2DAddress(16, m, n)], curSad);
                        cur1Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY1 + m), s_orgMb[Index2DAddress(16, m, n)], cur1Sad);
                        cur2Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY2 + m), s_orgMb[Index2DAddress(16, m, n)], cur2Sad);
                        cur3Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY3 + m), s_orgMb[Index2DAddress(16, m, n)], cur3Sad);
                        cur4Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY4 + m), s_orgMb[Index2DAddress(16, m, n)], cur4Sad);
                        cur5Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY5 + m), s_orgMb[Index2DAddress(16, m, n)], cur5Sad);
                        cur6Sad = __usad(tex2D(tex_ref, pucCurX + n, pucCurY6 + m), s_orgMb[Index2DAddress(16, m, n)], cur6Sad);
                }
        }

        
        short distance = 0;   
        if(curSad > cur1Sad) { distance = P1; curSad = cur1Sad;}
        if(curSad > cur2Sad) { distance = P2; curSad = cur2Sad;}
        if(curSad > cur3Sad) { distance = P3; curSad = cur3Sad;}
        if(curSad > cur4Sad) { distance = P4; curSad = cur4Sad;}
        if(curSad > cur5Sad) { distance = P5; curSad = cur5Sad;}
        if(curSad > cur6Sad) { distance = P6; curSad = cur6Sad;}
        
        mvY += distance;

       
        for(int s = STEP_SIZE; s > 0; s >>= 1){

               
                pucCurY = pucCurY + distance;
                cur1Sad = 0;
                cur2Sad = 0;
        
                pucCurY1  = pucCurY + (-s);
                pucCurY2  = pucCurY + s;

               
                for(int m = 0; m < 16; m++){
                        for(int n = 0; n < 16; n++){
                                
                                cur1Sad  = __usad(tex2D(tex_ref, pucCurX + n, pucCurY1 + m), s_orgMb[Index2DAddress(16, m, n)], cur1Sad);
                                cur2Sad  = __usad(tex2D(tex_ref, pucCurX + n, pucCurY2 + m), s_orgMb[Index2DAddress(16, m, n)], cur2Sad);
                                
                        }
                }

                distance = 0;   
                if(curSad > cur1Sad) { distance = -s; curSad = cur1Sad;}
                if(curSad > cur2Sad) { distance = s; curSad = cur2Sad;}
                mvY += distance;          

        }
        
        s_mv[tid].y = mvY;
        s_sad[tid] = curSad;

        s_mv[tid].x = tid - SEARCH_RANGE;
                                
                        
#if SEARCH_RANGE > 32
        __syncthreads();

        if(tid < 64){
                if(s_sad[tid] > s_sad[tid + 64]){
                        s_sad[tid] = s_sad[tid + 64];
                        s_mv[tid].y = s_mv[tid + 64].y;
                        s_mv[tid].x = s_mv[tid + 64].x;
                }
        }
#endif
        __syncthreads();
        
        if(tid < 32){   //in 1 warp, needless __syncthreads()   
                if(s_sad[tid] > s_sad[tid + 32]){
                        s_sad[tid] = s_sad[tid + 32];
                        s_mv[tid].y = s_mv[tid + 32].y;
                        s_mv[tid].x = s_mv[tid + 32].x;
                }
                if(s_sad[tid] > s_sad[tid + 16]){
                        s_sad[tid] = s_sad[tid + 16];
                        s_mv[tid].y = s_mv[tid + 16].y;
                        s_mv[tid].x = s_mv[tid + 16].x;
                }
                if(s_sad[tid] > s_sad[tid + 8]){
                        s_sad[tid] = s_sad[tid + 8];
                        s_mv[tid].y = s_mv[tid + 8].y;
                        s_mv[tid].x = s_mv[tid + 8].x;
                }
                if(s_sad[tid] > s_sad[tid + 4]){
                        s_sad[tid] = s_sad[tid + 4];
                        s_mv[tid].y = s_mv[tid + 4].y;
                        s_mv[tid].x = s_mv[tid + 4].x;
                }
                if(s_sad[tid] > s_sad[tid + 2]){
                        s_sad[tid] = s_sad[tid + 2];
                        s_mv[tid].y = s_mv[tid + 2].y;
                        s_mv[tid].x = s_mv[tid + 2].x;
                }
                if(s_sad[0] > s_sad[1]){
                        d_sad[by*(IMAGE_WIDTH/16) + bx] = s_sad[1];
                        d_mvY[by*(IMAGE_WIDTH/16) + bx] = s_mv[1].y;
                        d_mvX[by*(IMAGE_WIDTH/16) + bx] = s_mv[1].x;
                }else{
                        d_sad[by*(IMAGE_WIDTH/16) + bx] = s_sad[0];
                        d_mvY[by*(IMAGE_WIDTH/16) + bx] = s_mv[0].y;
                        d_mvX[by*(IMAGE_WIDTH/16) + bx] = s_mv[0].x;
                }
        }
}
//---------------------------------------------------------------------------------------------------
__global__ void kerneltest(int refStride, short* yRef, short* result1, short* result2){
        uint tid = threadIdx.x;
        uint bx  = blockIdx.x;
        uint by  = blockIdx.y;
        
        short* pucCur = yRef;
        short pucCurX = SEARCH_RANGE;
        short pucCurY = SEARCH_RANGE;
        
      
        //pucOrg = yOrg + Shift2Next16x16Block(orgStride, by, bx);
        //pucCur = yRef + Shift2Next16x16Block(refStride, by, bx);
        //pucCurX += bx*16;
        //pucCurY += by*16;
        pucCur += (-SEARCH_RANGE);
        pucCurX += (-SEARCH_RANGE);

#if 1    
        if(bx == 0 && by == 0){
                if(tid == 0){
                        for(int i = 0; i < 128; i++){
                                result1[i] = tex2D(tex_ref, pucCurX +i, pucCurY);
                                result2[i] = pucCur[Index2DAddress(refStride, 0, i)];
                        }
                }
        }
#endif
}
//---------------------------------------------------------------------------------------------------
void eggCudaPmodsMalloc(){

        int refStride = externEggEstimation.m_refStride;
       
        pucOrgStart = externEggEstimation.m_pYOrg;                                        
        pucRefStart = externEggEstimation.m_pYRef - Index2DAddress( refStride,
                                                                    SEARCH_RANGE,
                                                                    SEARCH_RANGE);
                                                
        refImageSize = (IMAGE_HEIGHT + SEARCH_RANGE*2)*
                       (IMAGE_WIDTH  + SEARCH_RANGE)*sizeof(short);
        
        orgImageSize = (IMAGE_HEIGHT + SEARCH_RANGE)*(IMAGE_WIDTH + SEARCH_RANGE)*sizeof(short);
        
        
        cutilSafeCall( hipMalloc( (void**) &d_yOrg, orgImageSize));
        
        cutilSafeCall( hipMemcpy( d_yOrg, pucOrgStart, orgImageSize, hipMemcpyHostToDevice) );

        cutilSafeCall( hipMallocArray( &cu_array_ref, &channelDesc, 
                                        (refStride),
                                        (IMAGE_HEIGHT + SEARCH_RANGE*2) ) );

          
        cutilSafeCall( hipMemcpyToArray(   cu_array_ref, 0, 0, pucRefStart, 
                                            refImageSize, hipMemcpyHostToDevice ) );

        cutilSafeCall( hipBindTextureToArray( tex_ref, cu_array_ref ));

        
}

//---------------------------------------------------------------------------------------------------
void eggCudaPmodsFree(){

        cutilSafeCall( hipFree(d_yOrg) );
        cutilSafeCall( hipUnbindTexture(tex_ref) );
        cutilSafeCall( hipFreeArray(cu_array_ref) );
}
//---------------------------------------------------------------------------------------------------
//parallel multithread one-dimensional search
void eggCudaPmods4x4Mv(mvMode_t &r2ListMv, blockMode_t &r2ListBlock){
       
        short*  h_mvX = new short[(IMAGE_WIDTH/4)*(IMAGE_HEIGHT/4)];
        short*  h_mvY = new short[(IMAGE_WIDTH/4)*(IMAGE_HEIGHT/4)];
        uint*   h_sad = new uint[(IMAGE_WIDTH/4)*(IMAGE_HEIGHT/4)];
        
        short*  d_mvX;
        short*  d_mvY;
        uint*   d_sad;

        cutilSafeCall(hipMalloc((void**)&d_mvX,(IMAGE_WIDTH/4)*(IMAGE_HEIGHT/4)*sizeof(short)));
        cutilSafeCall(hipMalloc((void**)&d_mvY,(IMAGE_WIDTH/4)*(IMAGE_HEIGHT/4)*sizeof(short)));
        cutilSafeCall(hipMalloc((void**)&d_sad,(IMAGE_WIDTH/4)*(IMAGE_HEIGHT/4)*sizeof(uint)));

        //hipEvent_t start, stop;
        //cutilSafeCall(hipEventCreate(&start));
        //cutilSafeCall(hipEventCreate(&stop));
        
        //hipEventRecord(start, 0);                                        
        kernelPmods4x4Mv<<<grid4x4Mv, BLOCK_DIM>>>( externEggEstimation.m_orgStride,
                                                    d_yOrg, d_mvX, d_mvY, d_sad);
        // check for any errors
        cutilCheckMsg("kernelPmods4x4Mv execution failed");
        //hipEventRecord(stop, 0);
/*
        while(hipEventQuery(stop) == hipErrorNotReady){
                
        }
*/
        
        cutilSafeCall( hipMemcpy( h_mvX, d_mvX, (IMAGE_WIDTH/4)*(IMAGE_HEIGHT/4)*sizeof(short),
                                   hipMemcpyDeviceToHost) );
        cutilSafeCall( hipMemcpy( h_mvY, d_mvY, (IMAGE_WIDTH/4)*(IMAGE_HEIGHT/4)*sizeof(short),
                                   hipMemcpyDeviceToHost) );
        cutilSafeCall( hipMemcpy( h_sad, d_sad, (IMAGE_WIDTH/4)*(IMAGE_HEIGHT/4)*sizeof(uint),
                                   hipMemcpyDeviceToHost) );

        for(int i = 0; i < IMAGE_HEIGHT/4; i++){
        	for(int j = 0; j < IMAGE_WIDTH/4; j++){
                        r2ListMv.mv4x4[i][j].x   = h_mvX[i*(IMAGE_WIDTH/4) + j];
                        r2ListMv.mv4x4[i][j].y   = h_mvY[i*(IMAGE_WIDTH/4) + j];
                        r2ListBlock.sad4x4[i][j] = h_sad[i*(IMAGE_WIDTH/4) + j];
                }
        }
        
        cutilSafeCall(hipFree(d_mvX));
        cutilSafeCall(hipFree(d_mvY));
        cutilSafeCall(hipFree(d_sad));

        delete[] h_mvX;
        delete[] h_mvY;
        delete[] h_sad;
        
}
//---------------------------------------------------------------------------------------------------
//parallel multithread one-dimensional search
void eggCudaPmods4x8Mv(mvMode_t &r2ListMv, blockMode_t &r2ListBlock){
                                               
        
        short*  h_mvX = new short[(IMAGE_WIDTH/4)*(IMAGE_HEIGHT/8)];
        short*  h_mvY = new short[(IMAGE_WIDTH/4)*(IMAGE_HEIGHT/8)];
        uint*   h_sad = new uint[(IMAGE_WIDTH/4)*(IMAGE_HEIGHT/8)];
        short*  d_mvX;
        short*  d_mvY;
        uint*   d_sad;

        cutilSafeCall(hipMalloc((void**)&d_mvX,(IMAGE_WIDTH/4)*(IMAGE_HEIGHT/8)*sizeof(short)));
        cutilSafeCall(hipMalloc((void**)&d_mvY,(IMAGE_WIDTH/4)*(IMAGE_HEIGHT/8)*sizeof(short)));
        cutilSafeCall(hipMalloc((void**)&d_sad,(IMAGE_WIDTH/4)*(IMAGE_HEIGHT/8)*sizeof(uint)));

                                             
        kernelPmods4x8Mv<<<grid4x8Mv, BLOCK_DIM>>>( externEggEstimation.m_orgStride,
                                                    d_yOrg, d_mvX, d_mvY, d_sad);
        // check for any errors
        cutilCheckMsg("kernelPmods4x8Mv execution failed");
        
        cutilSafeCall( hipMemcpy( h_mvX, d_mvX, (IMAGE_WIDTH/4)*(IMAGE_HEIGHT/8)*sizeof(short),
                                   hipMemcpyDeviceToHost) );
        cutilSafeCall( hipMemcpy( h_mvY, d_mvY, (IMAGE_WIDTH/4)*(IMAGE_HEIGHT/8)*sizeof(short),
                                   hipMemcpyDeviceToHost) );
        cutilSafeCall( hipMemcpy( h_sad, d_sad, (IMAGE_WIDTH/4)*(IMAGE_HEIGHT/8)*sizeof(uint),
                                   hipMemcpyDeviceToHost) );

        for(int i = 0; i < IMAGE_HEIGHT/8; i++){
        	for(int j = 0; j < IMAGE_WIDTH/4; j++){
                        r2ListMv.mv4x8[i][j].x   = h_mvX[i*(IMAGE_WIDTH/4) + j];
                        r2ListMv.mv4x8[i][j].y   = h_mvY[i*(IMAGE_WIDTH/4) + j];
                        r2ListBlock.sad4x8[i][j] = h_sad[i*(IMAGE_WIDTH/4) + j];
                }
        }
        
        cutilSafeCall(hipFree(d_mvX));
        cutilSafeCall(hipFree(d_mvY));
        cutilSafeCall(hipFree(d_sad));

        delete[] h_mvX;
        delete[] h_mvY;
        delete[] h_sad;
        
}
//---------------------------------------------------------------------------------------------------
//parallel multithread one-dimensional search
void eggCudaPmods8x4Mv(mvMode_t &r2ListMv, blockMode_t &r2ListBlock){
                                               
        
        short*  h_mvX = new short[(IMAGE_WIDTH/8)*(IMAGE_HEIGHT/4)];
        short*  h_mvY = new short[(IMAGE_WIDTH/8)*(IMAGE_HEIGHT/4)];
        uint*   h_sad = new uint[(IMAGE_WIDTH/8)*(IMAGE_HEIGHT/4)];
        short*  d_mvX;
        short*  d_mvY;
        uint*   d_sad;

        cutilSafeCall(hipMalloc((void**)&d_mvX,(IMAGE_WIDTH/8)*(IMAGE_HEIGHT/4)*sizeof(short)));
        cutilSafeCall(hipMalloc((void**)&d_mvY,(IMAGE_WIDTH/8)*(IMAGE_HEIGHT/4)*sizeof(short)));
        cutilSafeCall(hipMalloc((void**)&d_sad,(IMAGE_WIDTH/8)*(IMAGE_HEIGHT/4)*sizeof(uint)));

                                           
        kernelPmods8x4Mv<<<grid8x4Mv, BLOCK_DIM>>>( externEggEstimation.m_orgStride,
                                                    d_yOrg, d_mvX, d_mvY, d_sad);
        // check for any errors
        cutilCheckMsg("kernelPmods8x4Mv execution failed");
        
        
        cutilSafeCall( hipMemcpy( h_mvX, d_mvX, (IMAGE_WIDTH/8)*(IMAGE_HEIGHT/4)*sizeof(short),
                                   hipMemcpyDeviceToHost) );
        cutilSafeCall( hipMemcpy( h_mvY, d_mvY, (IMAGE_WIDTH/8)*(IMAGE_HEIGHT/4)*sizeof(short),
                                   hipMemcpyDeviceToHost) );
        cutilSafeCall( hipMemcpy( h_sad, d_sad, (IMAGE_WIDTH/8)*(IMAGE_HEIGHT/4)*sizeof(uint),
                                   hipMemcpyDeviceToHost) );

        for(int i = 0; i < IMAGE_HEIGHT/4; i++){
        	for(int j = 0; j < IMAGE_WIDTH/8; j++){
                        r2ListMv.mv8x4[i][j].x   = h_mvX[i*(IMAGE_WIDTH/8) + j];
                        r2ListMv.mv8x4[i][j].y   = h_mvY[i*(IMAGE_WIDTH/8) + j];
                        r2ListBlock.sad8x4[i][j] = h_sad[i*(IMAGE_WIDTH/8) + j];
                }
        }
        
        cutilSafeCall(hipFree(d_mvX));
        cutilSafeCall(hipFree(d_mvY));
        cutilSafeCall(hipFree(d_sad));

        delete[] h_mvX;
        delete[] h_mvY;
        delete[] h_sad;
        
}
//---------------------------------------------------------------------------------------------------
//parallel multithread one-dimensional search
void eggCudaPmods8x8Mv(mvMode_t &r2ListMv, blockMode_t &r2ListBlock){
                                                
        
        short*  h_mvX = new short[(IMAGE_WIDTH/8)*(IMAGE_HEIGHT/8)];
        short*  h_mvY = new short[(IMAGE_WIDTH/8)*(IMAGE_HEIGHT/8)];
        uint*   h_sad = new uint[(IMAGE_WIDTH/8)*(IMAGE_HEIGHT/8)];
        short*  d_mvX;
        short*  d_mvY;
        uint*   d_sad;

        cutilSafeCall(hipMalloc((void**)&d_mvX,(IMAGE_WIDTH/8)*(IMAGE_HEIGHT/8)*sizeof(short)));
        cutilSafeCall(hipMalloc((void**)&d_mvY,(IMAGE_WIDTH/8)*(IMAGE_HEIGHT/8)*sizeof(short)));
        cutilSafeCall(hipMalloc((void**)&d_sad,(IMAGE_WIDTH/8)*(IMAGE_HEIGHT/8)*sizeof(uint)));

                                               
        kernelPmods8x8Mv<<<grid8x8Mv, BLOCK_DIM>>>( externEggEstimation.m_orgStride,
                                                    d_yOrg, d_mvX, d_mvY, d_sad);
        // check for any errors
        cutilCheckMsg("kernelPmods8x8Mv execution failed");
        
        
        cutilSafeCall( hipMemcpy( h_mvX, d_mvX, (IMAGE_WIDTH/8)*(IMAGE_HEIGHT/8)*sizeof(short),
                                   hipMemcpyDeviceToHost) );
        cutilSafeCall( hipMemcpy( h_mvY, d_mvY, (IMAGE_WIDTH/8)*(IMAGE_HEIGHT/8)*sizeof(short),
                                   hipMemcpyDeviceToHost) );
        cutilSafeCall( hipMemcpy( h_sad, d_sad, (IMAGE_WIDTH/8)*(IMAGE_HEIGHT/8)*sizeof(uint),
                                   hipMemcpyDeviceToHost) );

        for(int i = 0; i < IMAGE_HEIGHT/8; i++){
        	for(int j = 0; j < IMAGE_WIDTH/8; j++){
                        r2ListMv.mv8x8[i][j].x   = h_mvX[i*(IMAGE_WIDTH/8) + j];
                        r2ListMv.mv8x8[i][j].y   = h_mvY[i*(IMAGE_WIDTH/8) + j];
                        r2ListBlock.sad8x8[i][j] = h_sad[i*(IMAGE_WIDTH/8) + j];
                }
        }
        
        cutilSafeCall(hipFree(d_mvX));
        cutilSafeCall(hipFree(d_mvY));
        cutilSafeCall(hipFree(d_sad));

        delete[] h_mvX;
        delete[] h_mvY;
        delete[] h_sad;
        
}
//---------------------------------------------------------------------------------------------------
//parallel multithread one-dimensional search
void eggCudaPmods8x16Mv(mvMode_t &r2ListMv, blockMode_t &r2ListBlock){
                                               
        
        short*  h_mvX = new short[(IMAGE_WIDTH/8)*(IMAGE_HEIGHT/16)];
        short*  h_mvY = new short[(IMAGE_WIDTH/8)*(IMAGE_HEIGHT/16)];
        uint*   h_sad = new uint[(IMAGE_WIDTH/8)*(IMAGE_HEIGHT/16)];
        short*  d_mvX;
        short*  d_mvY;
        uint*   d_sad;

        cutilSafeCall(hipMalloc((void**)&d_mvX,(IMAGE_WIDTH/8)*(IMAGE_HEIGHT/16)*sizeof(short)));
        cutilSafeCall(hipMalloc((void**)&d_mvY,(IMAGE_WIDTH/8)*(IMAGE_HEIGHT/16)*sizeof(short)));
        cutilSafeCall(hipMalloc((void**)&d_sad,(IMAGE_WIDTH/8)*(IMAGE_HEIGHT/16)*sizeof(uint)));

                                              
        kernelPmods8x16Mv<<<grid8x16Mv, BLOCK_DIM>>>( externEggEstimation.m_orgStride,
                                                      d_yOrg, d_mvX, d_mvY, d_sad);
        // check for any errors
        cutilCheckMsg("kernelPmods8x16Mv execution failed");
        
        
        cutilSafeCall( hipMemcpy( h_mvX, d_mvX, (IMAGE_WIDTH/8)*(IMAGE_HEIGHT/16)*sizeof(short),
                                   hipMemcpyDeviceToHost) );
        cutilSafeCall( hipMemcpy( h_mvY, d_mvY, (IMAGE_WIDTH/8)*(IMAGE_HEIGHT/16)*sizeof(short),
                                   hipMemcpyDeviceToHost) );
        cutilSafeCall( hipMemcpy( h_sad, d_sad, (IMAGE_WIDTH/8)*(IMAGE_HEIGHT/16)*sizeof(uint),
                                   hipMemcpyDeviceToHost) );

        for(int i = 0; i < IMAGE_HEIGHT/16; i++){
        	for(int j = 0; j < IMAGE_WIDTH/8; j++){
                        r2ListMv.mv8x16[i][j].x   = h_mvX[i*(IMAGE_WIDTH/8) + j];
                        r2ListMv.mv8x16[i][j].y   = h_mvY[i*(IMAGE_WIDTH/8) + j];
                        r2ListBlock.sad8x16[i][j] = h_sad[i*(IMAGE_WIDTH/8) + j];
                }
        }
        
        cutilSafeCall(hipFree(d_mvX));
        cutilSafeCall(hipFree(d_mvY));
        cutilSafeCall(hipFree(d_sad));

        delete[] h_mvX;
        delete[] h_mvY;
        delete[] h_sad;
        
}
//---------------------------------------------------------------------------------------------------
//parallel multithread one-dimensional search
void eggCudaPmods16x8Mv(mvMode_t &r2ListMv, blockMode_t &r2ListBlock){
       
        
        short*  h_mvX = new short[(IMAGE_WIDTH/16)*(IMAGE_HEIGHT/8)];
        short*  h_mvY = new short[(IMAGE_WIDTH/16)*(IMAGE_HEIGHT/8)];
        uint*   h_sad = new uint[(IMAGE_WIDTH/16)*(IMAGE_HEIGHT/8)];
        short*  d_mvX;
        short*  d_mvY;
        uint*   d_sad;

        cutilSafeCall(hipMalloc((void**)&d_mvX,(IMAGE_WIDTH/16)*(IMAGE_HEIGHT/8)*sizeof(short)));
        cutilSafeCall(hipMalloc((void**)&d_mvY,(IMAGE_WIDTH/16)*(IMAGE_HEIGHT/8)*sizeof(short)));
        cutilSafeCall(hipMalloc((void**)&d_sad,(IMAGE_WIDTH/16)*(IMAGE_HEIGHT/8)*sizeof(uint)));

                                               
        kernelPmods16x8Mv<<<grid16x8Mv, BLOCK_DIM>>>( externEggEstimation.m_orgStride,
                                                      d_yOrg, d_mvX, d_mvY, d_sad);
        // check for any errors
        cutilCheckMsg("kernelPmods16x8Mv execution failed");
        
        
        cutilSafeCall( hipMemcpy( h_mvX, d_mvX, (IMAGE_WIDTH/16)*(IMAGE_HEIGHT/8)*sizeof(short),
                                   hipMemcpyDeviceToHost) );
        cutilSafeCall( hipMemcpy( h_mvY, d_mvY, (IMAGE_WIDTH/16)*(IMAGE_HEIGHT/8)*sizeof(short),
                                   hipMemcpyDeviceToHost) );
        cutilSafeCall( hipMemcpy( h_sad, d_sad, (IMAGE_WIDTH/16)*(IMAGE_HEIGHT/8)*sizeof(uint),
                                   hipMemcpyDeviceToHost) );

        for(int i = 0; i < IMAGE_HEIGHT/8; i++){
        	for(int j = 0; j < IMAGE_WIDTH/16; j++){
                        r2ListMv.mv16x8[i][j].x   = h_mvX[i*(IMAGE_WIDTH/16) + j];
                        r2ListMv.mv16x8[i][j].y   = h_mvY[i*(IMAGE_WIDTH/16) + j];
                        r2ListBlock.sad16x8[i][j] = h_sad[i*(IMAGE_WIDTH/16) + j];
                }
        }
        
        cutilSafeCall(hipFree(d_mvX));
        cutilSafeCall(hipFree(d_mvY));
        cutilSafeCall(hipFree(d_sad));

        delete[] h_mvX;
        delete[] h_mvY;
        delete[] h_sad;
}

//---------------------------------------------------------------------------------------------------
//parallel multithread one-dimensional search
void eggCudaPmods16x16Mv(mvMode_t &r2ListMv, blockMode_t &r2ListBlock){
                                              
        short*  h_mvX = new short[(IMAGE_WIDTH/16)*(IMAGE_HEIGHT/16)];
        short*  h_mvY = new short[(IMAGE_WIDTH/16)*(IMAGE_HEIGHT/16)];
        uint*   h_sad = new uint[(IMAGE_WIDTH/16)*(IMAGE_HEIGHT/16)];
        
        short*  d_mvX;
        short*  d_mvY;
        uint*   d_sad;
        
        
        cutilSafeCall( hipMalloc((void**)&d_mvX,(IMAGE_WIDTH/16)*(IMAGE_HEIGHT/16)*sizeof(short)));
        cutilSafeCall( hipMalloc((void**)&d_mvY,(IMAGE_WIDTH/16)*(IMAGE_HEIGHT/16)*sizeof(short)));
        cutilSafeCall( hipMalloc((void**)&d_sad,(IMAGE_WIDTH/16)*(IMAGE_HEIGHT/16)*sizeof(uint)));
        

        kernelPmods16x16Mv<<<grid16x16Mv, BLOCK_DIM>>>( externEggEstimation.m_orgStride,
                                                        d_yOrg, d_mvX, d_mvY, d_sad);
        // check for any errors
        cutilCheckMsg("kernelPmods16x16Mv execution failed");

        
        cutilSafeCall( hipMemcpy( h_mvX, d_mvX, (IMAGE_WIDTH/16)*(IMAGE_HEIGHT/16)*sizeof(short),
                                   hipMemcpyDeviceToHost) );
       
        cutilSafeCall( hipMemcpy( h_mvY, d_mvY, (IMAGE_WIDTH/16)*(IMAGE_HEIGHT/16)*sizeof(short),
                                   hipMemcpyDeviceToHost) );
        cutilSafeCall( hipMemcpy( h_sad, d_sad, (IMAGE_WIDTH/16)*(IMAGE_HEIGHT/16)*sizeof(uint),
                                   hipMemcpyDeviceToHost) );
        
        
        for(int i = 0; i < IMAGE_HEIGHT/16; i++){
        	for(int j = 0; j < IMAGE_WIDTH/16; j++){
                        r2ListMv.mv16x16[i][j].x   = h_mvX[i*(IMAGE_WIDTH/16) + j];
                        r2ListMv.mv16x16[i][j].y   = h_mvY[i*(IMAGE_WIDTH/16) + j];
                        r2ListBlock.sad16x16[i][j] = h_sad[i*(IMAGE_WIDTH/16) + j];
                }
        }

        
        cutilSafeCall(hipFree(d_mvX));
        cutilSafeCall(hipFree(d_mvY));
        cutilSafeCall(hipFree(d_sad));
        
        delete[] h_mvX;
        delete[] h_mvY;
        delete[] h_sad;
}

